#include "hip/hip_runtime.h"
//
//
//
#include "MACException.h"
#include "Convolutional_CUDA.cuh"
#include "Activations.h"
//
#define THREADSPERBLOCK 1024
////
//// Note that any atomic operation can be implemented based on atomicCAS() (Compare And Swap). For example, atomicAdd() for double-precision floating-point numbers is not available on devices with compute capability lower than 6.0 but it can be implemented as follows: 
//#if __CUDA_ARCH__ < 600
//__device__ double atomicAdd_patch(double* address, double val)
//{
//    unsigned long long int* address_as_ull =
//                              (unsigned long long int*)address;
//    unsigned long long int old = *address_as_ull, assumed;
//
//    do {
//        assumed = old;
//        old = atomicCAS(address_as_ull, assumed,
//                        __double_as_longlong(val +
//                               __longlong_as_double(assumed)));
//
//    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
//    } while (assumed != old);
//
//    return __longlong_as_double(old);
//}
//#endif
////
////
//__global__ void
//test_cuda( double **tempo, MAC::Mapping* Map, int mod )
//{
//  printf("image(%d) = %f \n", mod, tempo[mod][421578]);
//  printf("Map(%d) = [%d,%d,%d] \n", 421578, Map[421578].x_, Map[421578].y_, Map[421578].z_);
//}
//
//
//
__global__ void
test_cuda()
{
#if __CUDA_ARCH__ >= 200
  printf("Just a test");
#endif
}
//
//
//
__global__ void fill_with_zeros( int Image_size, double* Image )
{
  //
  //
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  //
  if ( idx <  Image_size )
    Image[idx] = 0.;
}
//
//
//
__global__ void nabla_reset_cuda( double** Nabla )
{
  //
  // feature: blockIdx
  // weight: threadIdx
  Nabla[ blockDim.x ][ threadIdx.x ] = 0.;
}
__global__ void cuda_hello()
{
  printf("Hello World from GPU!\n");
}
//
//
//
template< typename Activate >
__global__ void
convolution_cuda( int      Num_features_in,
		  int      Feature_out,
		  int      Image_size_out,
		  int      Number_of_weights,
		  double** To_conv,
		  double*  Conv,
		  double*  Activation_map,
		  double*  Delta_map,
		  double** Shared_weights,
		  double*  Shared_biases,
		  int*     Weights_pos_oi )
{
  //
  //
  int odx = blockDim.x * blockIdx.x + threadIdx.x;
  //
  if ( odx <  Image_size_out )
    {
      Conv[odx]      = 0.;
      Delta_map[odx] = 0.;
      //
      double   conv = 0.;
      Activate activation;
      //
      for ( int feature = 0 ; feature < Num_features_in; feature++ )
	for ( int k = 0 ; k < Number_of_weights ; k++ )
	  {
	    int idx = Weights_pos_oi[ k + odx * Number_of_weights ];
	    conv += Shared_weights[Feature_out][k] * To_conv[feature][idx];
	  }
      //
      Conv[odx]      = activation.f(  conv + Shared_biases[Feature_out] );
      Delta_map[odx] = activation.df( conv + Shared_biases[Feature_out] );
    }
}
//
// Backward
//
template< typename Activate >
__global__ void
dW_x_f_cuda( int      Num_features,
	     int      Image_size_out,
	     int      Number_of_weights,
	     double** DW_x_f,
	     double** F_map,
	     int*     Weights_pos_oi )
{
  //
  //
  int odx = blockDim.x * blockIdx.x + threadIdx.x;
  //
  if ( odx <  Image_size_out )
    {
      //
      for ( int feature = 0 ; feature < Num_features; feature++ )
	for ( int k = 0 ; k < Number_of_weights ; k++ )
	  {
	    int idx           = Weights_pos_oi[ k + odx * Number_of_weights ];
	    int kernel_weight = feature*Number_of_weights + k;
	    DW_x_f[kernel_weight][odx] = F_map[feature][idx];
	  }
    }
}
//
//
//
template< typename Activate >
__global__ void
transpose_convolution_cuda( int      Num_features_in,
			    int      Feature_out,
			    int      Image_size_out,
			    int      Number_of_weights,
			    double** To_deconv,
			    double*  Deconv,
			    double*  Activation_map,
			    double*  Delta_map,
			    double** Shared_weights,
			    double*  Shared_biases,
			    int*     Weights_pos_io )
{
  //
  //
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  //
  if ( idx <  Image_size_out )
    {
      //
      Deconv[idx]    = 0.;
      Delta_map[idx] = 0.;
      //
      double deconv = 0.;
      Deconv[idx]   = 0.;
      Activate activation;
      //
      for ( int feature = 0 ; feature < Num_features_in; feature++ )
	for ( int k = 0 ; k < Number_of_weights ; k++ )
	  {
	    int odx = Weights_pos_io[ k + idx * Number_of_weights ];
	    if ( odx != 999999999 )
	      deconv += Shared_weights[feature][k] * To_deconv[feature][odx];
	  }
      //
      Deconv[idx]    = activation.f(  deconv + Shared_biases[Feature_out] );
      Delta_map[idx] = activation.df( deconv + Shared_biases[Feature_out] );
    }
}
//
// Backward
//
__global__ void
dWT_x_f_cuda( int      Feature,
	      int      Image_size_out,
	      int      Number_of_weights,
	      double** DWT_x_f,
	      double*  F_map,
	      int*     Weights_pos_io )
{
  //
  //
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  //
  if ( idx <  Image_size_out )
    {
      //
      for ( int k = 0 ; k < Number_of_weights ; k++ )
	{
	  DWT_x_f[k][idx] = 0.;
	  int odx = Weights_pos_io[ k + idx * Number_of_weights ];
	  //
	  if ( odx != 999999999 )
	    DWT_x_f[k][idx] = F_map[odx];
	}
    }
}
//
// Backward
//
template< typename Activate >
__global__ void
delta_feature_cuda( int      Num_features,
		    int      Image_size_out,
		    int      Number_of_weights,
		    double** Shared_weights,
		    double** Delta_y,
		    double** Delta_h,
		    double** Df,
		    int*     Weights_pos_oi )
{
  //
  //
  int odx = blockDim.x * blockIdx.x + threadIdx.x;
  //
  if ( odx <  Image_size_out )
    {
      //
      for ( int feature = 0 ; feature < Num_features; feature++ )
	{
	  double w_delta_y = 0.;
	  for ( int k = 0 ; k < Number_of_weights ; k++ )
	    {
	      int idx = Weights_pos_oi[ k + odx * Number_of_weights ];
	      w_delta_y += Shared_weights[feature][k] * Delta_y[0][idx];
	    }

	  //
	  Delta_h[feature][odx] = w_delta_y * Df[feature][odx];
	}
    }
}
//
// Backward
//
__global__ void
nabla_cuda( int      Feature,
	    int      Image_size_out,
	    int      Number_of_weights,
	    double** DWf,
	    double*  Delta_map,
	    double** Nabla_E_weights,
	    double*  Nabla_E_biases )
{
  //
  //
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  //
  if ( idx <  Image_size_out )
    {
      //
      double delta = Delta_map[idx];
      for ( int k = 0 ; k < Number_of_weights ; k++ )
	Nabla_E_weights[Feature][k] += delta * DWf[k][idx];
      //
      Nabla_E_biases[Feature]       += delta;
    }
}
//
//
//
__host__ 
MAC::Convolutional_CUDA::Convolutional_CUDA()
{}
//
//
//
__host__ void
MAC::Convolutional_CUDA::load_convolution_kernels(// features
						  const std::size_t   Num_of_features_in,
						  const std::size_t   Num_of_features_out,
						  // weights
						  const int           Number_of_weights,
						  double**            Shared_weights,
						  double*             Shared_biases,
						  // Weights position and transposed matrix
						  std::size_t         Im_size_in,
						  std::size_t         Im_size_out,
						  std::size_t**       Weights_pos_oi,
						  std::size_t**       Weights_pos_io )
{
  //
  // Initialization
  std::cout << "Convolutional_CUDA -- Load convolution kernels." << std::endl;
  // check on th device
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  //
  // 1. Allocation space on the GPU
  // Features
  number_of_features_in_  = Num_of_features_in;
  number_of_features_out_ = Num_of_features_out;
  // weights
  number_of_weights_      = Number_of_weights;
  im_size_in_             = Im_size_in;
  im_size_out_            = Im_size_out;
  // free the the device
  err = hipDeviceReset();
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  //
  // 1.1. Start allocation the weights and matrices
  err = hipMalloc( (void **)&d_shared_weights_, Num_of_features_out * sizeof(double*) );
  err = hipMalloc( (void **)&d_shared_biases_,  Num_of_features_out * sizeof(double) );
  // Weights position and transposed matrix
  err = hipMalloc( (void **)&d_weights_pos_oi_, Im_size_out * Number_of_weights * sizeof(int) );
  err = hipMalloc( (void **)&d_weights_pos_io_, Im_size_in  * Number_of_weights * sizeof(int) );
  if (err != hipSuccess)
    {
      fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  // weights
  for ( std::size_t p = 0 ; p < Num_of_features_out ; p++)
    {
      double *temp_weights;
      hipMalloc((void **)&temp_weights, Number_of_weights * sizeof(double) );
      // create a master pointer we will move into the pointer to pointer
      hipMemcpy(temp_weights, Shared_weights[p], Number_of_weights * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(&d_shared_weights_[p], &temp_weights, sizeof(double*), hipMemcpyHostToDevice);
      if (err != hipSuccess)
	{
	  fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
	  exit(EXIT_FAILURE);
	}
    }
  // Biases
  hipMemcpy( d_shared_biases_, Shared_biases, Num_of_features_out * sizeof(double), hipMemcpyHostToDevice );
  // Weights position
  int* weights_pos_oi = new int[Im_size_out * Number_of_weights];
  int* weights_pos_io = new int[Im_size_in  * Number_of_weights];
  //
  for ( std::size_t o = 0 ; o < Im_size_out ; o++ )
    for ( int k = 0 ; k < Number_of_weights ; k++ )
      {
	size_t odx          = k + o * Number_of_weights;
	weights_pos_oi[odx] = static_cast< int>( Weights_pos_oi[o][k] );
      }
  //
  for ( std::size_t i = 0 ; i < Im_size_in ; i++ )
    for ( int k = 0 ; k < Number_of_weights ; k++ )
      {
	size_t idx          = k + i * Number_of_weights;
	weights_pos_io[idx] = static_cast< int>( Weights_pos_io[i][k] );
      }
  //
  err = hipMemcpy( d_weights_pos_oi_, weights_pos_oi, 
		    Im_size_out * Number_of_weights * sizeof(int), hipMemcpyHostToDevice );
  err = hipMemcpy( d_weights_pos_io_, weights_pos_io, 
		    Im_size_in  * Number_of_weights * sizeof(int), hipMemcpyHostToDevice );
  if (err != hipSuccess)
    {
      fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  //
  // 1.2. Allocating nabla
  err = hipMalloc( (void **)&d_nabla_E_weights_, Num_of_features_out * sizeof(double*) );
  err = hipMalloc( (void **)&d_nabla_E_biases_,  Num_of_features_out * sizeof(double) );
  // weights
  for ( std::size_t p = 0 ; p < Num_of_features_out ; p++)
    {
      double *temp_weights;
      hipMalloc((void **)&temp_weights, Number_of_weights * sizeof(double) );
      // create a master pointer we will move into the pointer to pointer
      hipMemcpy(&d_nabla_E_weights_[p], &temp_weights, sizeof(double*), hipMemcpyHostToDevice);
      if (err != hipSuccess)
	{
	  fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
	  exit(EXIT_FAILURE);
	}
    }
  // reset nabla
  nabla_reset_cuda<<< Num_of_features_out, number_of_weights_ >>>( d_nabla_E_weights_ ); 
  fill_with_zeros<<< Num_of_features_out, 1 >>>( Num_of_features_out, d_nabla_E_biases_ );
};
//
//
//
__host__ void
MAC::Convolutional_CUDA::load_deconvolution_kernels(// features
						    const std::size_t   Num_of_features_in,
						    const std::size_t   Num_of_features_out,
						    // weights
						    const int           Number_of_weights,
						    double**            Shared_weights,
						    double*             Shared_biases,
						    // Weights position and transposed matrix
						    std::size_t         Im_size_in,
						    std::size_t         Im_size_out,
						    std::size_t**       Weights_pos_oi,
						    std::size_t**       Weights_pos_io )
{
  //
  // Initialization
  std::cout << "Convolutional_CUDA -- Load deconvolution kernels." << std::endl;
  // check on th device
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  //
  // 1. Allocation space on the GPU
  // Features
  number_of_features_in_  = Num_of_features_in;
  number_of_features_out_ = Num_of_features_out;
  // weights
  number_of_weights_      = Number_of_weights;
  im_size_in_             = Im_size_in;
  im_size_out_            = Im_size_out;
  // free the the device
  err = hipDeviceReset();
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  //
  // 1.1. Start allocation the weights and matrices
  err = hipMalloc((void **)&d_shared_weights_, Num_of_features_in * sizeof(double*) );
  err = hipMalloc((void **)&d_shared_biases_,  Num_of_features_out * sizeof(double) );
  // Weights position and transposed matrix
  err = hipMalloc((void **)&d_weights_pos_oi_, Im_size_out * Number_of_weights * sizeof(int) );
  err = hipMalloc((void **)&d_weights_pos_io_, Im_size_in  * Number_of_weights * sizeof(int) );
  if (err != hipSuccess)
    {
      fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  // weights
  for ( std::size_t p = 0 ; p < Num_of_features_in ; p++)
    {
      double *temp_weights;
      hipMalloc((void **)&temp_weights, Number_of_weights * sizeof(double) );
      // create a master pointer we will move into the pointer to pointer
      hipMemcpy(temp_weights, Shared_weights[p], Number_of_weights * sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(&d_shared_weights_[p], &temp_weights, sizeof(double*), hipMemcpyHostToDevice);
      if (err != hipSuccess)
	{
	  fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
	  exit(EXIT_FAILURE);
	}
    }
  // Biases
  hipMemcpy( d_shared_biases_, Shared_biases, Num_of_features_out * sizeof(double), hipMemcpyHostToDevice );
  // Weights position
  int* weights_pos_oi = new int[Im_size_out * Number_of_weights];
  int* weights_pos_io = new int[Im_size_in  * Number_of_weights];
  //
  for ( std::size_t o = 0 ; o < Im_size_out ; o++ )
    for ( int k = 0 ; k < Number_of_weights ; k++ )
      {
	size_t odx          = k + o * Number_of_weights;
	weights_pos_oi[odx] = static_cast< int>( Weights_pos_oi[o][k] );
      }
  //
  for ( std::size_t i = 0 ; i < Im_size_in ; i++ )
    for ( int k = 0 ; k < Number_of_weights ; k++ )
      {
	size_t idx          = k + i * Number_of_weights;
	weights_pos_io[idx] = static_cast< int>( Weights_pos_io[i][k] );
      }
  //
  err = hipMemcpy( d_weights_pos_oi_, weights_pos_oi, 
		    Im_size_out * Number_of_weights * sizeof(int), hipMemcpyHostToDevice );
  err = hipMemcpy( d_weights_pos_io_, weights_pos_io, 
		    Im_size_in  * Number_of_weights * sizeof(int), hipMemcpyHostToDevice );
  if (err != hipSuccess)
    {
      fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  //
  // 1.2. Allocating nabla
  err = hipMalloc( (void **)&d_nabla_E_weights_, Num_of_features_in * sizeof(double*) );
  err = hipMalloc( (void **)&d_nabla_E_biases_,  Num_of_features_out * sizeof(double) );
  // weights
  for ( std::size_t p = 0 ; p < Num_of_features_in ; p++)
    {
      double *temp_weights;
      hipMalloc((void **)&temp_weights, Number_of_weights * sizeof(double) );
      // create a master pointer we will move into the pointer to pointer
      hipMemcpy(&d_nabla_E_weights_[p], &temp_weights, sizeof(double*), hipMemcpyHostToDevice);
      if (err != hipSuccess)
	{
	  fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
	  exit(EXIT_FAILURE);
	}
    }
  // reset nabla
  nabla_reset_cuda<<< Num_of_features_in, number_of_weights_ >>>( d_nabla_E_weights_ );
  fill_with_zeros<<< Num_of_features_out, 1 >>>( Num_of_features_out, d_nabla_E_biases_ );
};
//
//
//
__host__ void
MAC::Convolutional_CUDA::load_feature_maps( double** Prev_feature_maps )
{
  //
  // Initialization
  std::cout << "Convolutional_CUDA -- Load previouse feature maps & prepare next maps." << std::endl;
  // check on th device
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  //
  // 1. Allocation space for the previous features on the GPU
  err = hipMalloc( (void **)&d_previouse_feature_maps_,
		    number_of_features_in_ * sizeof(double*) );

  for ( std::size_t p = 0 ; p < number_of_features_in_ ; p++)
    {
      double *temp_feature;
      hipMalloc((void **)&temp_feature, im_size_in_ * sizeof(double) );
      // create a master pointer we will move into the pointer to pointer
      hipMemcpy( temp_feature, Prev_feature_maps[p],
		  im_size_in_ * sizeof(double), hipMemcpyHostToDevice );
      //
      hipMemcpy(&d_previouse_feature_maps_[p], &temp_feature,
		 sizeof(double*), hipMemcpyHostToDevice);
      if (err != hipSuccess)
	{
	  fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
	  exit(EXIT_FAILURE);
	}
    }

  //
  // 2. Allocation space for the next features on the GPU
  err = hipMalloc( (void **)&d_next_feature_maps_,    im_size_out_ * sizeof(double) );
  err = hipMalloc( (void **)&d_next_activation_maps_, im_size_out_ * sizeof(double) );
  err = hipMalloc( (void **)&d_next_delta_maps_,      im_size_out_ * sizeof(double) );
  err = hipMalloc( (void **)&d_target_maps_,          im_size_out_ * sizeof(double) );

  //
  // 3. Fill the next features with zeros
  int threadsPerBlock = THREADSPERBLOCK;
  int Blocks_out      = (( im_size_out_ ) + threadsPerBlock - 1) / threadsPerBlock;
  fill_with_zeros<<< Blocks_out, threadsPerBlock >>>( im_size_out_,
						      d_next_feature_maps_ );
  fill_with_zeros<<< Blocks_out, threadsPerBlock >>>( im_size_out_,
						      d_next_activation_maps_ );
  fill_with_zeros<<< Blocks_out, threadsPerBlock >>>( im_size_out_,
						      d_next_delta_maps_ );
  fill_with_zeros<<< Blocks_out, threadsPerBlock >>>( im_size_out_, d_target_maps_ );
}
//
//
//
__host__ void
MAC::Convolutional_CUDA::convolution( double**         Next_feature_maps,
				      double**         Next_activation_maps,
				      double**         Next_delta_maps,
				      const Functions& Activation_func )
{
  std::cout << "Convolutional_CUDA -- Run the convolution." << std::endl;
  //
  // 1. check on the device  
  int threadsPerBlock = THREADSPERBLOCK;
  int numBlocks       = (( im_size_out_ ) + threadsPerBlock - 1) / threadsPerBlock;
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  //
  // 2. Convolution and move the maps
  switch( Activation_func.get_function_name() )
    {
    case Func::F_TANH:
      {
	for ( std::size_t feature = 0 ; feature < number_of_features_out_; feature++ )
	  {
	    // 2.1. convolution
	    convolution_cuda< MAC::Activation_tanh ><<< numBlocks, threadsPerBlock >>>
	      ( static_cast< int >(number_of_features_in_),
		static_cast< int >(feature),
		static_cast< int >(im_size_out_),
		number_of_weights_,
		d_previouse_feature_maps_,
		d_next_feature_maps_,
		d_next_activation_maps_,
		d_next_delta_maps_,
		d_shared_weights_, d_shared_biases_,
		d_weights_pos_oi_ );
	    // 2.2 move the feature map back
	    hipMemcpy( Next_feature_maps[feature], d_next_feature_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
	    hipMemcpy( Next_activation_maps[feature], d_next_activation_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
	    hipMemcpy( Next_delta_maps[feature], d_next_delta_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
//	    // 2.3 reset the feature map
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_feature_maps_ );
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_activation_maps_ );
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_delta_maps_ );
	  }
	break;
      }
    case Func::F_SIGMOID:
      {
	for ( std::size_t feature = 0 ; feature < number_of_features_out_; feature++ )
	  {
	    // 2.1. convolution
	    convolution_cuda< MAC::Activation_sigmoid ><<< numBlocks, threadsPerBlock >>>
	      ( static_cast< int >(number_of_features_in_),
		static_cast< int >(feature),
		static_cast< int >(im_size_out_),
		number_of_weights_,
		d_previouse_feature_maps_,
		d_next_feature_maps_,
		d_next_activation_maps_,
		d_next_delta_maps_,
		d_shared_weights_, d_shared_biases_,
		d_weights_pos_oi_ );
	    // 2.2 move the feature map back
	    hipMemcpy( Next_feature_maps[feature], d_next_feature_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
	    hipMemcpy( Next_activation_maps[feature], d_next_activation_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
	    hipMemcpy( Next_delta_maps[feature], d_next_delta_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
//	    // 2.3 reset the feature map
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_feature_maps_ );
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_activation_maps_ );
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_delta_maps_ );
	  }
	break;
      }
    case Func::UNDETERMINED:
    default:
      {
	fprintf(stderr, "Wrong activation function.\n");
	exit(EXIT_FAILURE);
      }
    }
}
//
//
//
__host__ void
MAC::Convolutional_CUDA::transpose_convolution( double**         Next_feature_maps,
						double**         Next_activation_maps,
						double**         Next_delta_maps,
						const Functions& Activation_func )
{
  std::cout << "Convolutional_CUDA -- Run the deconvolution." << std::endl;
  //
  // 1. check on the device and load the 
  int threadsPerBlock = THREADSPERBLOCK;
  int numBlocks       = (( im_size_out_ ) + threadsPerBlock - 1) / threadsPerBlock;
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  
  //
  // 2. Convolution and move the maps
  switch( Activation_func.get_function_name() )
    {
    case Func::F_TANH:
      {
	for ( std::size_t feature = 0 ; feature < number_of_features_out_; feature++ )
	  {
	    // 2.1. deconvolution
	    transpose_convolution_cuda< MAC::Activation_tanh ><<< numBlocks, threadsPerBlock >>>
	      ( static_cast< int >(number_of_features_in_),
		static_cast< int >(feature),
		static_cast< int >(im_size_out_),
		number_of_weights_,
		d_previouse_feature_maps_,
		d_next_feature_maps_,
		d_next_activation_maps_,
		d_next_delta_maps_,
		d_shared_weights_, d_shared_biases_,
		d_weights_pos_oi_ );
	    // 2.2 move the feature map back
	    hipMemcpy( Next_feature_maps[feature], d_next_feature_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
	    hipMemcpy( Next_activation_maps[feature], d_next_activation_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
	    hipMemcpy( Next_delta_maps[feature], d_next_delta_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
//	    // 2.3 reset the feature map
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_feature_maps_ );
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_activation_maps_ );
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_delta_maps_ );
	  }
	break;
      }
    case Func::F_SIGMOID:
      {
	for ( std::size_t feature = 0 ; feature < number_of_features_out_; feature++ )
	  {
	    // 2.1. deconvolution
	    transpose_convolution_cuda< MAC::Activation_sigmoid ><<< numBlocks, threadsPerBlock >>>
	      ( static_cast< int >(number_of_features_in_),
		static_cast< int >(feature),
		static_cast< int >(im_size_out_),
		number_of_weights_,
		d_previouse_feature_maps_,
		d_next_feature_maps_,
		d_next_activation_maps_,
		d_next_delta_maps_,
		d_shared_weights_, d_shared_biases_,
		d_weights_pos_oi_ );
	    // 2.2 move the feature map back
	    hipMemcpy( Next_feature_maps[feature], d_next_feature_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
	    hipMemcpy( Next_activation_maps[feature], d_next_activation_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
	    hipMemcpy( Next_delta_maps[feature], d_next_delta_maps_,
			im_size_out_ * sizeof(double), hipMemcpyDeviceToHost );
//	    // 2.3 reset the feature map
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_feature_maps_ );
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_activation_maps_ );
//	    fill_with_zeros<<< numBlocks, threadsPerBlock >>>
//	      ( im_size_out_, d_next_delta_maps_ );
	  }
	break;
      }
    case Func::UNDETERMINED:
    default:
      {
	fprintf(stderr, "Wrong activation function.\n");
	exit(EXIT_FAILURE);
      }
    }
}
//
//
//
__host__ void
MAC::Convolutional_CUDA::backprog_transpose_convolution( double** Delta,
							 double** Features,
							 double** Nabla_w, 
							 double*  Nabla_b )
{
  std::cout << "Convolutional_CUDA -- Run backprog_transpose_convolution." << std::endl;
  std::cout
    << "\n number_of_features_in_ "  << number_of_features_in_
    << "\n number_of_features_out_ " << number_of_features_out_ 
    << "\n number_of_weights_ " << number_of_weights_ 
    << "\n im_size_in_ " << im_size_in_ 
    << "\n im_size_out_ " <<  im_size_out_
    << std::endl;
  //
  // 1. check on the device  
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  //
  // 2. create space on the device
  // 2.1. allocate dWxh vectors
  err = hipMalloc( (void **)&d_next_feature_maps_, im_size_out_       * sizeof(double) );
  err = hipMalloc( (void **)&d_next_delta_maps_,   im_size_in_        * sizeof(double) );
  err = hipMalloc( (void **)&d_dWT_x_f_,           number_of_weights_ * sizeof(double*) );
  //
  for ( int p = 0 ; p < number_of_weights_ ; p++)
    {
      double *temp;
      hipMalloc((void **)&temp, im_size_out_ * sizeof(double) );
      // 
      hipMemcpy(&d_dWT_x_f_[p], &temp, sizeof(double*), hipMemcpyHostToDevice);
      if (err != hipSuccess)
	{
	  fprintf(stderr, "error on the CUDA device (error code %s)!\n", hipGetErrorString(err));
	  exit(EXIT_FAILURE);
	}
    }


  //
  // 3. gradient descent 
  for ( std::size_t feature = 0 ; feature < number_of_features_in_; feature++ )
    {
      //
      // 3.1. Copy the feature information
      hipMemcpy( d_next_feature_maps_, Features[feature], 
		  im_size_out_ * sizeof(double), hipMemcpyHostToDevice );
      //
      // 3.2. dWxh vectors
      int threadsPerBlock = THREADSPERBLOCK;
      int numBlocks       = (( im_size_in_ ) + threadsPerBlock - 1) / threadsPerBlock;
      //
      dWT_x_f_cuda<<< numBlocks, threadsPerBlock >>>
	( static_cast< int >(feature),
	  static_cast< int >(im_size_in_),
	  number_of_weights_,
	  d_dWT_x_f_,
	  d_next_feature_maps_,
	  d_weights_pos_io_ );
      // 3.2. Compute nabla
      threadsPerBlock = THREADSPERBLOCK;
      numBlocks       = (( im_size_out_ ) + threadsPerBlock - 1) / threadsPerBlock;
      //
      for ( std::size_t s = 0 ; s < number_of_features_out_ ; s++ )
	{
	  //
	  // 3.2.1. Copy the feature information
	  hipMemcpy(d_next_delta_maps_ , Delta[s], 
		      im_size_out_ * sizeof(double), hipMemcpyHostToDevice );
	  // 3.2.2. nabla
	  nabla_cuda<<< numBlocks, threadsPerBlock >>>
	    ( static_cast< int >(feature),
	      static_cast< int >(im_size_out_),
	      number_of_weights_,
	      d_dWT_x_f_, d_next_delta_maps_,
	      d_nabla_E_weights_, d_nabla_E_biases_ );
	}
    }

  //
  // 4. free dW^Txh
}
//
//
//
__host__ void
MAC::Convolutional_CUDA::forward()
{
  std::cout << "Go fwd CUDA" << std::endl;
};
//
//
//
__host__ void
MAC::Convolutional_CUDA::backward( std::map< std::string, Neurons_type >& Neurons,
				   const Functions& Activation_func )
{
  std::cout << "Go bckwd CUDA" << std::endl;
};
//
//
//
__host__ 
MAC::Convolutional_CUDA::~Convolutional_CUDA()
{
  hipError_t err = hipGetLastError();
  err = hipFree( d_shared_weights_ );
  err = hipFree( d_shared_biases_ );
  err = hipFree( d_weights_pos_oi_ );
  err = hipFree( d_weights_pos_io_ );
  err = hipFree( d_previouse_feature_maps_ );
  err = hipFree( d_next_feature_maps_ );
  //
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to free device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  // Reset the device and exit
  // hipDeviceReset causes the driver to clean up all states. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  err = hipDeviceReset();
  //
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
}
